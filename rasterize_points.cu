/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

std::tuple<torch::Tensor, torch::Tensor> getGaussianXy(
	torch::Tensor& means3D,
	torch::Tensor& viewmatrix,
	torch::Tensor& projmatrix, 
	const int image_height, 
	const int image_width)
{
	const int P = means3D.size(0);
	const int H = image_height;
	const int W = image_width;
	
	torch::Tensor xy = torch::full({P, 3}, 0.0, means3D.options().dtype(torch::kFloat32));				// ! [u, v, depth]
	torch::Tensor zero_tensor = torch::full({3, 2}, 0.0, means3D.options().dtype(torch::kFloat32));
	if(P!= 0)
	{
		CudaRasterizer::Rasterizer::getGaussianXy(
			P, 
			means3D.contiguous().data<float>(),
			viewmatrix.contiguous().data<float>(),
			projmatrix.contiguous().data<float>(), 
			xy.contiguous().data<float>(), 
			W, H);
	}
	return std::make_tuple(xy, zero_tensor);
}
